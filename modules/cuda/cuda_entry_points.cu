#include "hip/hip_runtime.h"
// K-3D
// Copyright (c) 1995-2008, Timothy M. Shead
//
// Contact: tshead@k-3d.com
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public
// License as published by the Free Software Foundation; either
// version 2 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
//
// You should have received a copy of the GNU General Public
// License along with this program; if not, write to the Free Software
// Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA

/** \file
    \author Evan Lezar (evanlezar@gmail.com)
*/

// cuda includes
#include <stdio.h>
#include <cutil.h>
#include <hip/hip_vector_types.h>
//include the kernels
#include "cuda_kernels.cu"

// define the externals
#include "cuda_entry_points.h"

/**
 * Initialize the timing info structure
 */
void initTimingInfo(timingInfo_t* tInfo, int numberOfEntries)
{
	(*tInfo).numEntries = numberOfEntries;
	(*tInfo).timings = (double*)malloc ( numberOfEntries*sizeof(double) );
	(*tInfo).labels = (char**)malloc ( numberOfEntries*sizeof(char*) );
	for ( int i = 0 ; i < numberOfEntries ; i++ )
	{
		(*tInfo).labels[i] = (char*) malloc ( 33*sizeof(char) );
	}	
} 

/**
 * Set the label of a given timing_info entry
 */
inline void setTimingInfoLabel(timingInfo_t* tInfo, int index, char* label)
{
	sprintf((*tInfo).labels[index], "%s", label);
}

/**
 * Set the timing value of a given timing_info entry
 */
inline void setTimingInfoTiming(timingInfo_t* tInfo, int index, double timingVal)
{
	(*tInfo).timings[index] = timingVal;
}

/**
 * Integer division and rounding up
 */
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void CUDA_initialize_device()
{
	// initialise the device
	CUT_DEVICE_INIT();
}

/// entry point for the CUDA version of the BitmapAdd BitmapSubtract and BitmapMultiply plugin
extern "C" void bitmap_arithmetic_kernel_entry(int operation, unsigned short* p_deviceImage, int width, int height, float value)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

	switch ( operation )
	{
    	case CUDA_BITMAP_ADD:
    		// execute the add
    		add_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    		break;
    	case CUDA_BITMAP_MULTIPLY:
    		// execute the multiply kernel
    		multiply_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    		break;
    	case CUDA_BITMAP_SUBTRACT:
    		// execute the add kernel with value negated
    		add_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, -value);
    		break;
        case CUDA_BITMAP_GAMMA:
            // execute the gamma kernel
            gamma_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    	default:
    		// unknown operation 
    		;
	}
		
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    // Make sure this function blocks until the calculation is complete
    hipDeviceSynchronize();
}

extern "C" void bitmap_color_monochrome_kernel_entry(unsigned short* p_deviceImage, int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	// allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));
	
	color_monochrome_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, redWeight, greenWeight, blueWeight);
    	
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    hipDeviceSynchronize();
	
}

extern "C" void copy_and_bind_texture_to_array( void** cudaArrayPointer, float* arrayData, int width, int height )
{
	// alocate a hipArray to store the transformation matrix
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* cu_array;
	CUDA_SAFE_CALL( hipMallocArray( &cu_array, &channelDesc, width, height )); 
    CUDA_SAFE_CALL( hipMemcpyToArray( cu_array, 0, 0, arrayData, width*height*sizeof(float), hipMemcpyHostToDevice));
	
	// set texture parameters
    transformTexture.addressMode[0] = hipAddressModeClamp;
    transformTexture.addressMode[1] = hipAddressModeClamp;
    transformTexture.filterMode = hipFilterModePoint;
    transformTexture.normalized = false;
	
	// Bind the array to the texture
    CUDA_SAFE_CALL( hipBindTextureToArray( transformTexture, cu_array, channelDesc));
	
	*cudaArrayPointer = (void*)cu_array;
}

extern "C" void free_CUDA_array ( void* cudaArrayPointer )
{
	CUDA_SAFE_CALL(hipFreeArray((hipArray*)cudaArrayPointer));			
}

extern "C" void apply_linear_transform_to_point_data ( float *device_points, float *T_matrix, int num_points )
{
	dim3 threads_per_block(64, 1);
    dim3 blocks_per_grid( iDivUp(num_points, 64), 1);
	
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)device_points, num_points);
	
	// check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    hipDeviceSynchronize();
}

extern "C" void allocate_device_memory ( void** device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMalloc(device_pointer, size_in_bytes));	
}

extern "C" void copy_from_host_to_device ( void* device_pointer, const void* host_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(device_pointer, host_pointer, size_in_bytes, hipMemcpyHostToDevice));	
}

extern "C" void copy_from_device_to_host ( void* host_pointer, const void* device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(host_pointer, device_pointer, size_in_bytes, hipMemcpyDeviceToHost));	
}

extern "C" void free_device_memory ( void* device_pointer )
{
	CUDA_SAFE_CALL(hipFree(device_pointer));			
}

extern "C" void allocate_pinned_host_memory ( void** pointer_on_host, size_t size_in_bytes )  
{
	CUDA_SAFE_CALL(hipHostMalloc(pointer_on_host, size_in_bytes));	
}

extern "C" void free_pinned_host_memory ( void* pointer_on_host )  
{
	CUDA_SAFE_CALL(hipHostFree(pointer_on_host));	
}

extern "C" void transform_points_synchronous ( double *InputPoints, double *PointSelection, double *OutputPoints, int num_points, timingInfo_t* tInfo )
{
	#define OTHER 0
	#define CONVERT_PRE 1
	#define TO_DEVICE 2
	#define EXECUTE 3
	#define TO_HOST 4
	#define CONVERT_POST 5
	
	// initialize the timing info structure
	initTimingInfo(tInfo, 6);
    
    setTimingInfoLabel(tInfo, OTHER, "OTHER");
	setTimingInfoLabel(tInfo, CONVERT_PRE, "CONVERT_PRE");
	setTimingInfoLabel(tInfo, TO_DEVICE, "TO_DEVICE");
	setTimingInfoLabel(tInfo, EXECUTE, "EXECUTE");
	setTimingInfoLabel(tInfo, TO_HOST, "TO_HOST");
	setTimingInfoLabel(tInfo, CONVERT_POST, "CONVERT_POST");
		
	
	// use a cuda timer for timing
	unsigned int timer = 0;
	CUT_SAFE_CALL( cutCreateTimer( &timer ) );
	
	CUT_SAFE_CALL( cutStartTimer( timer));
    float *device_points;
	
	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);	
	
	// allocate pinned host memory to allow for asynchronous operations	
	float *host_points_single_p;
	allocate_pinned_host_memory ((void**)&host_points_single_p, num_points*sizeof(float)*4);
	
	dim3 threads_per_block(64, 1);
	dim3 blocks_per_grid( iDivUp(num_points, 64), 1);
	
	CUT_SAFE_CALL( cutStopTimer( timer));
    setTimingInfoTiming(tInfo, OTHER, cutGetTimerValue( timer));
		
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	for (int point = 0; point < num_points; ++point)
	{
		int float_index = (point)*4;
		int double_index = (point)*3;
		host_points_single_p[float_index] = (float)InputPoints[double_index];
		host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
		host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
		host_points_single_p[float_index+3] = (float)PointSelection[point];
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, CONVERT_PRE, cutGetTimerValue( timer));	
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));		
	CUDA_SAFE_CALL ( hipMemcpy(device_points, host_points_single_p, num_points*16, hipMemcpyHostToDevice) );
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, TO_DEVICE, cutGetTimerValue( timer));
		
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)(device_points), num_points);
	hipDeviceSynchronize();
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, EXECUTE, cutGetTimerValue( timer));
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	CUDA_SAFE_CALL ( hipMemcpy(host_points_single_p, device_points, num_points*16, hipMemcpyDeviceToHost) );
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, TO_HOST, cutGetTimerValue( timer));
		
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	for (int point = 0; point < num_points; ++point)
	{
		int float_index = (point)*4;
		int double_index = (point)*3;
		OutputPoints[double_index] = host_points_single_p[float_index];
		OutputPoints[double_index+1] = host_points_single_p[float_index+1];
		OutputPoints[double_index+2] = host_points_single_p[float_index+2];
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, CONVERT_POST, cutGetTimerValue( timer));
		
	CUT_SAFE_CALL( cutResetTimer( timer));	
	CUT_SAFE_CALL( cutStartTimer( timer));
	free_device_memory(device_points);
	free_pinned_host_memory ( host_points_single_p );
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[OTHER] += cutGetTimerValue( timer);	
	CUT_SAFE_CALL ( cutDeleteTimer ( timer ));
}

extern "C" void transform_points_asynchronous ( double *InputPoints, double *PointSelection, double *OutputPoints, int num_points, timingInfo_t* tInfo )
{
	#define OTHER 0
	#define STREAM_CREATE 1
	#define PHASE_1 2
	#define PHASE_2 3
	#define STREAM_DESTROY 4
	
	// initialize the timing info structure
	initTimingInfo(tInfo, 5);
    
    setTimingInfoLabel(tInfo, OTHER, "OTHER");
	setTimingInfoLabel(tInfo, STREAM_CREATE, "STREAM_CREATE");
	setTimingInfoLabel(tInfo, PHASE_1, "CONVERT_TO_DEVICE_EXECUTE");
	setTimingInfoLabel(tInfo, PHASE_2, "TO_HOST_CONVERT");
	setTimingInfoLabel(tInfo, STREAM_DESTROY, "STREAM_DESTROY");
	
	// use a cuda timer for timing
	unsigned int timer = 0;
	CUT_SAFE_CALL( cutCreateTimer( &timer ) );
	
	CUT_SAFE_CALL( cutStartTimer( timer));
	// set the number of streams
	int nstreams = 4;
	
    float *device_points;
	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);	
	
	// allocate pinned host memory to allow for asynchronous operations	
	float *host_points_single_p;
	allocate_pinned_host_memory ((void**)&host_points_single_p, num_points*sizeof(float)*4);
	
	int points_per_stream = num_points/nstreams;

	dim3 threads_per_block(32, 1);
	dim3 blocks_per_grid( iDivUp(points_per_stream, 32), 1);
	
	CUT_SAFE_CALL( cutStopTimer( timer));
    setTimingInfoTiming(tInfo, OTHER, cutGetTimerValue( timer));
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	// allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int n = 0; n < nstreams; n++)
    	CUDA_SAFE_CALL( hipStreamCreate(&(streams[n])) ); 
	
	CUT_SAFE_CALL( cutStopTimer( timer));
    setTimingInfoTiming(tInfo, STREAM_CREATE, cutGetTimerValue( timer));
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	for ( int n = 0; n < nstreams; n++ )
	{
		// Convert a subset of the data to floats 
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			host_points_single_p[float_index] = (float)InputPoints[double_index];
			host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
			host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
			host_points_single_p[float_index+3] = (float)PointSelection[point];
		}
		
		// for each stream copy the data to the device and execute the kernel
		CUDA_SAFE_CALL ( hipMemcpyAsync(device_points + n*points_per_stream*4, host_points_single_p + n*points_per_stream*4, points_per_stream*16, hipMemcpyHostToDevice, streams[n]) );
		linear_transform_kernel <<< blocks_per_grid, threads_per_block, 0, streams[n] >>> ((float4*)(device_points + n*points_per_stream*4), points_per_stream);
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
    setTimingInfoTiming(tInfo, PHASE_1, cutGetTimerValue( timer));
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	// copy the data back from the device and convert
	for ( int n = 0; n < nstreams; n++ )
	{
		CUDA_SAFE_CALL ( hipMemcpyAsync(host_points_single_p + n*points_per_stream*4, device_points + n*points_per_stream*4, points_per_stream*16, hipMemcpyDeviceToHost, streams[n]) );
		// need to synchronize the streams so that the data is available to copy to the output points	
		hipStreamSynchronize(streams[n]);		
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			OutputPoints[double_index] = host_points_single_p[float_index];
			OutputPoints[double_index+1] = host_points_single_p[float_index+1];
			OutputPoints[double_index+2] = host_points_single_p[float_index+2];
		}		
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, PHASE_2, cutGetTimerValue( timer));
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	// release resources
	for(int n = 0; n < nstreams; n++)
	{
    	hipStreamDestroy(streams[n]);
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	setTimingInfoTiming(tInfo, STREAM_DESTROY, cutGetTimerValue( timer));
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	free_device_memory(device_points);
	free_pinned_host_memory ( host_points_single_p );
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[OTHER] += cutGetTimerValue( timer);	
	CUT_SAFE_CALL ( cutDeleteTimer ( timer ));
}

extern "C" void subdivide_edges_split_point_calculator ( unsigned int* edge_indices, 
                                                         unsigned int num_edge_indices, 
                                                         float* points_and_selection,
                                                         unsigned int num_points,  
                                                         unsigned int* edge_point_indices,
                                                         unsigned int* clockwise_edge_indices,
                                                         float* new_points_and_selection,
                                                         int num_split_points )
{
    int threads_x = 64 / num_split_points;
    
    dim3 threads_per_block(threads_x, num_split_points);
    dim3 blocks_per_grid( iDivUp(num_points, threads_x), 1);
    
    printf("CUDA DEBUG: Calling split kernel\n");
    
    subdivide_edges_split_point_kernel<<< blocks_per_grid, threads_per_block >>> ( edge_indices, 
                                                                                   num_edge_indices, 
                                                                                   (float4*)points_and_selection, 
                                                                                   num_points,
                                                                                   (float4*)new_points_and_selection, 
                                                                                   edge_point_indices, 
                                                                                   clockwise_edge_indices, 
                                                                                   num_split_points );  
    
    // check if the kernel executed correctly
    //CUT_CHECK_ERROR("Kernel execution failed");
    
    hipError_t last_error = hipGetLastError();
    
    printf("CUDA ERROR: %s\n", hipGetErrorString(last_error));
    
    hipDeviceSynchronize();
}

extern "C" void copy_2D_from_host_to_device_with_padding ( void* device_pointer, const void* host_pointer, size_t device_pitch, size_t host_pitch, size_t width_in_bytes, size_t rows )
{
    CUDA_SAFE_CALL ( hipMemcpy2D(device_pointer, device_pitch, host_pointer, host_pitch, width_in_bytes, rows, hipMemcpyHostToDevice) );
}

/**
 * Call thread synchronize to ensure consistency
 */
extern "C" void synchronize_threads ()
{
    hipDeviceSynchronize();    
}
