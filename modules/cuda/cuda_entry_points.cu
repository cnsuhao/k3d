#include "hip/hip_runtime.h"
// cuda includes
#include <stdio.h>
#include <cutil.h>
#include <hip/hip_vector_types.h>
//include the kernels
#include "cuda_kernels.cu"

// define the externals
#include "cuda_entry_points.h"

// pointer to the image in device memory
static ushort4 *d_image = 0;
unsigned int input_size = 0;

// integer division and rounding up
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void CUDA_initialize_device()
{
	// initialise the device
	CUT_DEVICE_INIT();
}
extern "C" void bitmap_copy_data_from_host_to_device(const unsigned short *input, int width, int height)
{
	// width and height is the number of pixels - each pixel is 4*16bits = 8bytes
	input_size = 8 * width * height;
      
    // allocate the memory on the device    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_image, input_size));

    // copy the data to the device
    CUDA_SAFE_CALL(hipMemcpy(d_image, input, input_size, hipMemcpyHostToDevice));	
	
}

/// entry point for the CUDA version of the BitmapAdd BitmapSubtract and BitmapMultiply plugin
extern "C" void bitmap_arithmetic_kernel_entry(int operation, int width, int height, float value)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

	switch ( operation )
	{
    	case CUDA_BITMAP_ADD:
    		// execute the add
    		add_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, value);
    		break;
    	case CUDA_BITMAP_MULTIPLY:
    		// execute the multiply kernel
    		multiply_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, value);
    		break;
    	case CUDA_BITMAP_SUBTRACT:
    		// execute the add kernel with value negated
    		add_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, -value);
    		break;
    	default:
    		// unknown operation
    		;
	}
		
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    // Make sure this function blocks until the calculation is complete
    hipDeviceSynchronize();
}

extern "C" void bitmap_color_monochrome_kernel_entry(int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	// allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));
	
	color_monochrome_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, redWeight, greenWeight, blueWeight);
    	
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    hipDeviceSynchronize();
	
}

extern "C" void bitmap_copy_data_from_device_to_host(unsigned short *output, int width, int height)
{
	// copy the results to the output image
    CUDA_SAFE_CALL(hipMemcpy(output, d_image, input_size, hipMemcpyDeviceToHost));
}

extern "C" void CUDA_cleanup()
{
	// cleanup memory
    CUDA_SAFE_CALL(hipFree(d_image));	
}

extern "C" void copy_and_bind_texture_to_array( void** cudaArrayPointer, float* arrayData, int width, int height )
{
	// alocate a hipArray to store the transformation matrix
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* cu_array;
	CUDA_SAFE_CALL( hipMallocArray( &cu_array, &channelDesc, width, height )); 
    CUDA_SAFE_CALL( hipMemcpyToArray( cu_array, 0, 0, arrayData, width*height*sizeof(float), hipMemcpyHostToDevice));
	
	// set texture parameters
    transformTexture.addressMode[0] = hipAddressModeClamp;
    transformTexture.addressMode[1] = hipAddressModeClamp;
    transformTexture.filterMode = hipFilterModePoint;
    transformTexture.normalized = false;
	
	// Bind the array to the texture
    CUDA_SAFE_CALL( hipBindTextureToArray( transformTexture, cu_array, channelDesc));
	
	*cudaArrayPointer = (void*)cu_array;
}

extern "C" void free_CUDA_array ( void* cudaArrayPointer )
{
	CUDA_SAFE_CALL(hipFreeArray((hipArray*)cudaArrayPointer));			
}

extern "C" void apply_linear_transform_to_point_data ( float *device_points, float *T_matrix, int num_points )
{
	dim3 threads_per_block(64, 1);
    dim3 blocks_per_grid( iDivUp(num_points, 64), 1);
	
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)device_points, num_points);
	
	// check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    hipDeviceSynchronize();
}

extern "C" void allocate_device_memory ( void** device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMalloc(device_pointer, size_in_bytes));	
}

extern "C" void copy_from_host_to_device ( void* device_pointer, const void* host_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(device_pointer, host_pointer, size_in_bytes, hipMemcpyHostToDevice));	
}

extern "C" void copy_from_device_to_host ( void* host_pointer, const void* device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(host_pointer, device_pointer, size_in_bytes, hipMemcpyDeviceToHost));	
}

extern "C" void free_cuda_pointer ( void* device_pointer )
{
	CUDA_SAFE_CALL(hipFree(device_pointer));			
}

extern "C" void allocate_pinned_host_memory ( void** pointer_on_host, size_t size_in_bytes )  
{
	CUDA_SAFE_CALL(hipHostMalloc(pointer_on_host, size_in_bytes));	
}

extern "C" void free_pinned_host_memory ( void* pointer_on_host )  
{
	CUDA_SAFE_CALL(hipHostFree(pointer_on_host));	
}

extern "C" void test_stream_implementation ( double *InputPoints, double *PointSelection, float* host_points_single_p, int num_points )
{
	// set the number of streams
	int nstreams = 4;
    float *device_points;
	
	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);	
		
		
	// allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
    	CUDA_SAFE_CALL( hipStreamCreate(&(streams[i])) ); 
	
	int points_per_stream = num_points/nstreams;
	
#ifdef __DEVICE_EMULATION__	
	printf("Num points : %d : Points per stream : %d\n", num_points, points_per_stream); 
	printf("device base : %x : host base : %x\n", device_points, host_points_single_p);
#endif
	
	dim3 threads_per_block(64, 1);
	dim3 blocks_per_grid( iDivUp(points_per_stream, 64), 1);
	
	for ( int n = 0; n < nstreams; n++ )
	{
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			host_points_single_p[float_index] = (float)InputPoints[double_index];
			host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
			host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
			host_points_single_p[float_index+3] = (float)PointSelection[point];
		}
		
#ifdef __DEVICE_EMULATION__		
		printf("device adr : %x : host adr : %x : num bytes : %d \n", device_points + n*points_per_stream*4, host_points_single_p + n*points_per_stream*4, points_per_stream*16);
#endif

		CUDA_SAFE_CALL ( hipMemcpyAsync(device_points + n*points_per_stream*4, host_points_single_p + n*points_per_stream*4, points_per_stream*16, hipMemcpyHostToDevice, streams[n]) );
		linear_transform_kernel <<< blocks_per_grid, threads_per_block, 0, streams[n] >>> ((float4*)(device_points + n*points_per_stream*4), points_per_stream);
		//linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)(device_points + n*points_per_stream*16), points_per_stream);
		CUDA_SAFE_CALL ( hipMemcpyAsync(host_points_single_p + n*points_per_stream*4, device_points + n*points_per_stream*4, points_per_stream*16, hipMemcpyDeviceToHost, streams[n]) );
				
	}
	//for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
	
	// release resources
	for(int i = 0; i < nstreams; i++)
	{
		hipStreamSynchronize(streams[i]);
    	hipStreamDestroy(streams[i]);
	}
	
	free_cuda_pointer(device_points);
}

