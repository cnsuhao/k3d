#include "hip/hip_runtime.h"
// cuda includes
#include <stdio.h>
#include <cutil.h>
#include <hip/hip_vector_types.h>
//include the kernels
#include "cuda_kernels.cu"

// define the externals
#include "cuda_entry_points.h"

// pointer to the image in device memory
static ushort4 *d_image = 0;
unsigned int input_size = 0;

// integer division and rounding up
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void CUDA_initialize_device()
{
	// initialise the device
	CUT_DEVICE_INIT();
}
extern "C" void bitmap_copy_data_from_host_to_device(const unsigned short *input, int width, int height)
{
	// width and height is the number of pixels - each pixel is 4*16bits = 8bytes
	input_size = 8 * width * height;
      
    // allocate the memory on the device    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_image, input_size));

    // copy the data to the device
    CUDA_SAFE_CALL(hipMemcpy(d_image, input, input_size, hipMemcpyHostToDevice));	
	
}

/// entry point for the CUDA version of the BitmapAdd BitmapSubtract and BitmapMultiply plugin
extern "C" void bitmap_arithmetic_kernel_entry(int operation, int width, int height, float value)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

	switch ( operation )
	{
    	case CUDA_BITMAP_ADD:
    		// execute the add
    		add_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, value);
    		break;
    	case CUDA_BITMAP_MULTIPLY:
    		// execute the multiply kernel
    		multiply_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, value);
    		break;
    	case CUDA_BITMAP_SUBTRACT:
    		// execute the add kernel with value negated
    		add_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, -value);
    		break
    		;
    	default:
    		// unknown operation
    		;
	}
		
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    // Make sure this function blocks until the calculation is complete
    hipDeviceSynchronize();
}

extern "C" void bitmap_color_monochrome_kernel_entry(int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	// allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));
	
	color_monochrome_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, redWeight, greenWeight, blueWeight);
    	
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
	
}

extern "C" void bitmap_copy_data_from_device_to_host(unsigned short *output, int width, int height)
{
	// copy the results to the output image
    CUDA_SAFE_CALL(hipMemcpy(output, d_image, input_size, hipMemcpyDeviceToHost));
}

extern "C" void CUDA_cleanup()
{
	// cleanup memory
    CUDA_SAFE_CALL(hipFree(d_image));	
}


extern "C" void apply_linear_transform_to_point_data ( float *device_points, float *device_matrix, int num_points )
{
	dim3 threads_per_block(64, 1);
    dim3 blocks_per_grid( iDivUp(num_points, 64), 1);
	
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)device_points, (float4*)device_matrix, num_points);
	
	// check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");					
}

extern "C" void test_double_to_float_entry ( double *in, float *out, int num )
{
	dim3 threads_per_block(8, 1);
    dim3 blocks_per_grid( iDivUp(num, 8), 1);
	
	test_double_to_float<<< blocks_per_grid, threads_per_block >>> (in, out, num);
	
	// check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");		
}

extern "C" void allocate_device_memory ( void** device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMalloc(device_pointer, size_in_bytes));	
}

extern "C" void copy_from_host_to_device ( void* device_pointer, const void* host_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(device_pointer, host_pointer, size_in_bytes, hipMemcpyHostToDevice));	
}

extern "C" void copy_from_device_to_host ( void* host_pointer, const void* device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(host_pointer, device_pointer, size_in_bytes, hipMemcpyDeviceToHost));	
}

extern "C" void free_cuda_pointer ( void* device_pointer )
{
	CUDA_SAFE_CALL(hipFree(device_pointer));			
}
