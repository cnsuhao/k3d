#include "hip/hip_runtime.h"
// cuda includes
#include <cutil.h>
#include <hip/hip_vector_types.h>
#include <stdio.h>
//include the kernels
#include "cuda_kernels.cu"

// define the externals
#include "cuda_entry_points.h"

// pointer to the image in device memory
static ushort4 *d_image = 0;
unsigned int input_size = 0;

// integer division and rounding up
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void CUDA_initialize_device()
{
	// initialise the device
	CUT_DEVICE_INIT();
}
extern "C" void bitmap_copy_data_from_host_to_device(const unsigned short *input, int width, int height)
{
	// width and height is the number of pixels - each pixel is 4*16bits = 8bytes
	input_size = 8 * width * height;
    
    // allocate the memory on the device    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_image, input_size));

    // copy the data to the device
    CUDA_SAFE_CALL(hipMemcpy(d_image, input, input_size, hipMemcpyHostToDevice));	
	
}

/// entry point for the CUDA version of the BitmapAdd plugin
extern "C" void bitmap_add_entry(int width, int height, float value)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

    // execute the kernel
    add_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, value);
    
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
}

extern "C" void bitmap_copy_data_from_device_to_host(unsigned short *output, int width, int height)
{
	// copy the results to the output image
    CUDA_SAFE_CALL(hipMemcpy(output, d_image, input_size, hipMemcpyDeviceToHost));
}

extern "C" void CUDA_cleanup()
{
	// cleanup memory
    CUDA_SAFE_CALL(hipFree(d_image));	
}