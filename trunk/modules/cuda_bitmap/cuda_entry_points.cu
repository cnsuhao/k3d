#include "hip/hip_runtime.h"
// cuda includes
#include <cutil.h>
#include <hip/hip_vector_types.h>
//include the kernels
#include "cuda_kernels.cu"

#include <stdio.h>

// forward declaration of the entry functions
extern "C" void bitmap_add_entry(const unsigned short *input, unsigned short *output, int width, int height, float value);

// integer division and rounding up
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void bitmap_add_entry(const unsigned short *input, unsigned short *output, int width, int height, float value)
{
	// initialise the device
	CUT_DEVICE_INIT();
	
 	// width and height is the number of pixels - each pixel is 4*16bits = 8bytes
	const unsigned int input_size = 8 * width * height;
    
    // pointer to the image on the device
    ushort4 *d_image;
    // allocate the memory on the device    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_image, input_size));
    
    // copy the data to the device
    CUDA_SAFE_CALL(hipMemcpy(d_image, input, input_size, hipMemcpyHostToDevice));
    
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

    // execute the kernel
    add_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, value);
    
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
	
	
	//CUDA_SAFE_CALL(hipMemcpy(output, input, input_size, hipMemcpyHostToHost));
	 // copy the results to the output image
    CUDA_SAFE_CALL(hipMemcpy(output, d_image, input_size, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_image));
	
    	
}

