#include "hip/hip_runtime.h"
#ifndef _CUDA_KERNELS_H_
#define _CUDA_KERNELS_H_

#ifdef __DEVICE_EMULATION__
#include <stdio.h>
#endif

// declare the texture reference for Matrix multiplication
texture<float, 2> transformTexture;

// convert a half-float to a single precision float
__device__ float halfToFloat (unsigned short halfIn)
{
	int s = (halfIn >> 15) & 0x00000001;
    int e = (halfIn >> 10) & 0x0000001f;
    int m =  halfIn        & 0x000003ff;
	
	int val = 0;
	int done = 0;
	
    if (e == 0)
    {
		if (m == 0)
		{
	    	//
	    	// Plus or minus zero
	    	//
			val = s << 31;
			done = 1;
		}
		else
		{
	    	//
	    	// Denormalized number -- renormalize it
	    	//

	    	while (!(m & 0x00000400))
	    	{
				m <<= 1;
				e -=  1;
	    	}

	    	e += 1;
	    	m &= ~0x00000400;
		}	
    }
    else if (e == 31)
    {
		if (m == 0)
		{
	    	//
	    	// Positive or negative infinity
	    	//

	    	val = (s << 31) | 0x7f800000;
	    	done = 1;
		}
		else
		{
	    	//
	    	// Nan -- preserve sign and significand bits
	    	//

	    	val = (s << 31) | 0x7f800000 | (m << 13);
	    	done = 1;
		}
    }

    //
    // Normalized number
    //
	if ( !done )
	{
		e = e + (127 - 15);
    	m = m << 13;

    	//
    	// Assemble s, e and m.
    	//
		val = (s << 31) | (e << 23) | m;
	}
	
	return __int_as_float (val);
		
}

// convert a single precision float to a half-float
__device__ unsigned short floatToHalf( float floatIn )
{
    //
    // Our floating point number, f, is represented by the bit
    // pattern in integer i.  Disassemble that bit pattern into
    // the sign, s, the exponent, e, and the significand, m.
    // Shift s into the position where it will go in in the
    // resulting half number.
    // Adjust e, accounting for the different exponent bias
    // of float and half (127 versus 15).
    //
	
	int floatBits = __float_as_int( floatIn );
	
    int s =  ((floatBits >> 16) & 0x00008000);
    int e = ((floatBits >> 23) & 0x000000ff) - (127 - 15);
    int m =   floatBits        & 0x007fffff;
    //
    // Now reassemble s, e and m into a half:
    //

    if (e <= 0)
    {
		if (e < -10)
		{
	    	//
	    	// E is less than -10.  The absolute value of f is
	    	// less than HALF_MIN (f may be a small normalized
	    	// float, a denormalized float or a zero).
	    	//
	    	// We convert f to a half zero.
	    	//

	    	return 0;
		}
		
		//
		// E is between -10 and 0.  F is a normalized float,
		// whose magnitude is less than HALF_NRM_MIN.
		//
		// We convert f to a denormalized half.
		// 

		m = (m | 0x00800000) >> (1 - e);

		//
		// Round to nearest, round "0.5" up.
		//
		// Rounding may cause the significand to overflow and make
		// our number normalized.  Because of the way a half's bits
		// are laid out, we don't have to treat this case separately;
		// the code below will handle it correctly.
		// 

		if (m &  0x00001000)
	    	m += 0x00002000;

		//
		// Assemble the half from s, e (zero) and m.
		//
		return s | (m >> 13);
    }
    else if (e == 0xff - (127 - 15))
    {
		if (m == 0)
		{
	    	//
	    	// F is an infinity; convert f to a half
	    	// infinity with the same sign as f.
	    	//
			return s | 0x7c00;
		}
		else
		{
	    	//
	    	// F is a NAN; produce a half NAN that preserves
	    	// the sign bit and the 10 leftmost bits of the
	    	// significand of f.
	    	//
			return s | 0x7c00 | (m >> 13);
		}
    }
    else
    {
		//
		// E is greater than zero.  F is a normalized float.
		// We try to convert f to a normalized half.
		//

		//
		// Round to nearest, round "0.5" up
		//

		if (m &  0x00001000)
		{
	    	m += 0x00002000;

	    	if (m & 0x00800000)
	    	{
				m =  0;		// overflow in significand,
				e += 1;		// adjust exponent
	    	}
		}

		//
		// Handle exponent overflow
		//

		if (e > 30)
		{
			return s | 0x7c00;	// if this returns, the half becomes an
	    	 // infinity with the same sign as f.
		}   							

		//
		// Assemble the half from s, e and m.
		//
		return s | (e << 10) | (m >> 13);
    }	
}

__global__ void add_kernel (ushort4 *image_RGBA, int width, int height, float value)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w         
        const int idx = width * iy + ix;
        
        float4 pixelFloat;
        
        pixelFloat.x = halfToFloat((unsigned short)image_RGBA[idx].x) + value;
        pixelFloat.y = halfToFloat((unsigned short)image_RGBA[idx].y) + value;
        pixelFloat.z = halfToFloat((unsigned short)image_RGBA[idx].z) + value;
        
        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
	__syncthreads();
} 

__global__ void multiply_kernel (ushort4 *image_RGBA, int width, int height, float value)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w         
        const int idx = width * iy + ix;
        
        float4 pixelFloat;
        
        pixelFloat.x = halfToFloat((unsigned short)image_RGBA[idx].x) * value;
        pixelFloat.y = halfToFloat((unsigned short)image_RGBA[idx].y) * value;
        pixelFloat.z = halfToFloat((unsigned short)image_RGBA[idx].z) * value;
        
        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
	__syncthreads();
}

/**
 * Apply gamma transform to each pixel value of an immage
 */
__global__ void gamma_kernel (ushort4 *image_RGBA, int width, int height, float value)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w         
        const int idx = width * iy + ix;
        
        float4 pixelFloat;
        
        pixelFloat.x = powf(halfToFloat((unsigned short)image_RGBA[idx].x), value);
        pixelFloat.y = powf(halfToFloat((unsigned short)image_RGBA[idx].y), value);
        pixelFloat.z = powf(halfToFloat((unsigned short)image_RGBA[idx].z), value);
        
        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
}

/**
 * Invert each color channel value:  out = max - in + min  with max = 1 and min = 0
 */
__global__ void invert_kernel (ushort4 *image_RGBA, int width, int height)
{
    #define MAX_HALF 1.0f
    
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w         
        const int idx = width * iy + ix;
        
        float4 pixelFloat;
        
        pixelFloat.x = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].x);
        pixelFloat.y = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].y);
        pixelFloat.z = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].z);
        
        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
}

__global__ void color_monochrome_kernel ( ushort4 *image_RGBA, int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w         
        const int idx = width * iy + ix;
        
        float monoValue;
        
        monoValue = halfToFloat(image_RGBA[idx].x) * redWeight 
        		  + halfToFloat(image_RGBA[idx].y) * greenWeight 
        		  + halfToFloat(image_RGBA[idx].z) * blueWeight;  
        
        image_RGBA[idx].x = floatToHalf(monoValue);
        image_RGBA[idx].y = floatToHalf(monoValue);
        image_RGBA[idx].z = floatToHalf(monoValue);
    }
	__syncthreads();
}

__global__ void linear_transform_kernel ( float4 *points, int num_points )
{
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	__shared__ float T[4][4];
	
	__shared__ float4 ThreadPoints[64];
	
	/*
	__syncthreads();
	
	if ( threadIdx.x < 1 )
	{
		T[0][0] = tex2D(transformTexture, 0, 0);
		T[1][0] = tex2D(transformTexture, 0, 1);
		T[2][0] = tex2D(transformTexture, 0, 2);
		T[3][0] = tex2D(transformTexture, 0, 3);
		T[0][1] = tex2D(transformTexture, 1, 0);
		T[1][1] = tex2D(transformTexture, 1, 1);
		T[2][1] = tex2D(transformTexture, 1, 2);
		T[3][1] = tex2D(transformTexture, 1, 3);
		T[0][2] = tex2D(transformTexture, 2, 0);
		T[1][2] = tex2D(transformTexture, 2, 1);
		T[2][2] = tex2D(transformTexture, 2, 2);
		T[3][2] = tex2D(transformTexture, 2, 3);
		T[0][3] = tex2D(transformTexture, 3, 0);
		T[1][3] = tex2D(transformTexture, 3, 1);
		T[2][3] = tex2D(transformTexture, 3, 2);
		T[3][3] = tex2D(transformTexture, 3, 3);
	}
	*/
	
    if ( threadIdx.x < 16 )
	{
		//const int row = idx >> 2;
		//const int col = idx & 0x3; 
		T[idx >> 2][idx & 0x3] = tex2D(transformTexture, idx & 0x3, idx >> 2);
	}
	if ( blockDim.x < 16 )
	{
		for ( int i = blockDim.x ; i < 16 ; i++ )
		{
			T[i >> 2][i & 0x3] = tex2D(transformTexture, i & 0x3, i >> 2);		
		}	
	}
	__syncthreads();
	
	if ( idx < num_points )
	{
		float4 vt;
		ThreadPoints[threadIdx.x] = points[idx];
		vt.w = T[3][0]*ThreadPoints[threadIdx.x].x + T[3][1]*ThreadPoints[threadIdx.x].y + T[3][2]*ThreadPoints[threadIdx.x].z + T[3][3];

		vt.x = (T[0][0]*ThreadPoints[threadIdx.x].x + T[0][1]*ThreadPoints[threadIdx.x].y + T[0][2]*ThreadPoints[threadIdx.x].z + T[0][3])/vt.w;
		vt.y = (T[1][0]*ThreadPoints[threadIdx.x].x + T[1][1]*ThreadPoints[threadIdx.x].y + T[1][2]*ThreadPoints[threadIdx.x].z + T[1][3])/vt.w;
		vt.z = (T[2][0]*ThreadPoints[threadIdx.x].x + T[2][1]*ThreadPoints[threadIdx.x].y + T[2][2]*ThreadPoints[threadIdx.x].z + T[2][3])/vt.w;
		
		points[idx].x = ThreadPoints[threadIdx.x].x*(1 - ThreadPoints[threadIdx.x].w) + ThreadPoints[threadIdx.x].w*vt.x;
		points[idx].y = ThreadPoints[threadIdx.x].y*(1 - ThreadPoints[threadIdx.x].w) + ThreadPoints[threadIdx.x].w*vt.y;
		points[idx].z = ThreadPoints[threadIdx.x].z*(1 - ThreadPoints[threadIdx.x].w) + ThreadPoints[threadIdx.x].w*vt.z;
	}
	
	__syncthreads();	

}

/**
 * Kernel for calculating the coordinates of the new points along the specified edges.
 * 
 */
__global__ void subdivide_edges_split_point_kernel ( unsigned int* edge_indices, 
                                                     unsigned int num_edge_indices, 
                                                     float4* points_and_selection,
                                                     unsigned int num_points,
                                                     float4* new_points_and_selection, 
                                                     unsigned int* edge_point_indices,
                                                     unsigned int* clockwise_edge_indices,
                                                     int num_split_points )
{
    
    unsigned int edge_index_index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int split_index = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    if ( edge_index_index < num_edge_indices )
    {
        
        unsigned int edge_index = edge_indices[edge_index_index];
        unsigned int p_index = edge_point_indices[edge_index];
        unsigned int new_point_index = edge_index_index*num_split_points + split_index;
        
        #ifdef __DEVICE_EMULATION__
            printf("Edge Index Index: %d\n", edge_index_index);
            printf("Split Index: %d\n", split_index);
            printf("Edge Index: %d\n", edge_index);
            printf("Clockwise Edge Index: %d\n", clockwise_edge_indices[edge_index]);
            printf("Point Index: %d\n", edge_point_indices[edge_index]);
            printf("Clockwise Point Index: %d\n", edge_point_indices[clockwise_edge_indices[edge_index]]);
            printf("New Point Index: %d\n", new_point_index);
        #endif 
        
        
        float4 p0 = points_and_selection[p_index];
        float4 p1 = points_and_selection[edge_point_indices[clockwise_edge_indices[edge_index]]];

        #ifdef __DEVICE_EMULATION__
            printf("P_0:%d: (%f, %f, %f)\n", p_index, p0.x, p0.y, p0.z);
            printf("P_1:%d: (%f, %f, %f)\n", edge_point_indices[clockwise_edge_indices[edge_index]], p1.x, p1.y, p1.z);
        #endif
        
        p1.x = (p1.x - p0.x) / (num_split_points + 1);
        p1.y = (p1.y - p0.y) / (num_split_points + 1);
        p1.z = (p1.z - p0.z) / (num_split_points + 1);
        
        #ifdef __DEVICE_EMULATION__
            printf("P_delta:(%f, %f, %f)\n", p1.x, p1.y, p1.z);
        #endif
        
        new_points_and_selection[new_point_index].x = p0.x + (split_index + 1)*p1.x;
        new_points_and_selection[new_point_index].y = p0.y + (split_index + 1)*p1.y;
        new_points_and_selection[new_point_index].z = p0.z + (split_index + 1)*p1.z;
        new_points_and_selection[new_point_index].w = 1;
        
    } 
      
}

#endif // #ifndef _CUDA_KERNELS_H_

