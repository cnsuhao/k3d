#include "hip/hip_runtime.h"
// cuda includes
#include <stdio.h>
#include <cutil.h>
#include <hip/hip_vector_types.h>
//include the kernels
#include "cuda_kernels.cu"

// define the externals
#include "cuda_entry_points.h"

// pointer to the image in device memory
static ushort4 *d_image = 0;
unsigned int input_size = 0;

/**
 * Integer division and rounding up
 */
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void CUDA_initialize_device()
{
	// initialise the device
	CUT_DEVICE_INIT();
}
extern "C" void bitmap_copy_data_from_host_to_device(const unsigned short *input, int width, int height)
{
	// width and height is the number of pixels - each pixel is 4*16bits = 8bytes
	input_size = 8 * width * height;
      
    // allocate the memory on the device    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_image, input_size));

    // copy the data to the device
    CUDA_SAFE_CALL(hipMemcpy(d_image, input, input_size, hipMemcpyHostToDevice));	
	
}

/// entry point for the CUDA version of the BitmapAdd BitmapSubtract and BitmapMultiply plugin
extern "C" void bitmap_arithmetic_kernel_entry(int operation, unsigned short* p_deviceImage, int width, int height, float value)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

	switch ( operation )
	{
    	case CUDA_BITMAP_ADD:
    		// execute the add
    		add_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    		break;
    	case CUDA_BITMAP_MULTIPLY:
    		// execute the multiply kernel
    		multiply_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    		break;
    	case CUDA_BITMAP_SUBTRACT:
    		// execute the add kernel with value negated
    		add_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)d_image, width, height, -value);
    		break;
    	default:
    		// unknown operation
    		;
	}
		
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    // Make sure this function blocks until the calculation is complete
    hipDeviceSynchronize();
}

extern "C" void bitmap_color_monochrome_kernel_entry(int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	// allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));
	
	color_monochrome_kernel<<< blocks_per_grid, threads_per_block >>> (d_image, width, height, redWeight, greenWeight, blueWeight);
    	
    // check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    hipDeviceSynchronize();
	
}

extern "C" void bitmap_copy_data_from_device_to_host(unsigned short *output, int width, int height)
{
	// copy the results to the output image
    CUDA_SAFE_CALL(hipMemcpy(output, d_image, input_size, hipMemcpyDeviceToHost));
}

extern "C" void CUDA_cleanup()
{
	// cleanup memory
    CUDA_SAFE_CALL(hipFree(d_image));	
}

extern "C" void copy_and_bind_texture_to_array( void** cudaArrayPointer, float* arrayData, int width, int height )
{
	// alocate a hipArray to store the transformation matrix
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* cu_array;
	CUDA_SAFE_CALL( hipMallocArray( &cu_array, &channelDesc, width, height )); 
    CUDA_SAFE_CALL( hipMemcpyToArray( cu_array, 0, 0, arrayData, width*height*sizeof(float), hipMemcpyHostToDevice));
	
	// set texture parameters
    transformTexture.addressMode[0] = hipAddressModeClamp;
    transformTexture.addressMode[1] = hipAddressModeClamp;
    transformTexture.filterMode = hipFilterModePoint;
    transformTexture.normalized = false;
	
	// Bind the array to the texture
    CUDA_SAFE_CALL( hipBindTextureToArray( transformTexture, cu_array, channelDesc));
	
	*cudaArrayPointer = (void*)cu_array;
}

extern "C" void free_CUDA_array ( void* cudaArrayPointer )
{
	CUDA_SAFE_CALL(hipFreeArray((hipArray*)cudaArrayPointer));			
}

extern "C" void apply_linear_transform_to_point_data ( float *device_points, float *T_matrix, int num_points )
{
	dim3 threads_per_block(64, 1);
    dim3 blocks_per_grid( iDivUp(num_points, 64), 1);
	
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)device_points, num_points);
	
	// check if the kernel executed correctly
    CUT_CHECK_ERROR("Add Kernel execution failed");
    hipDeviceSynchronize();
}

extern "C" void allocate_device_memory ( void** device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMalloc(device_pointer, size_in_bytes));	
}

extern "C" void copy_from_host_to_device ( void* device_pointer, const void* host_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(device_pointer, host_pointer, size_in_bytes, hipMemcpyHostToDevice));	
}

extern "C" void copy_from_device_to_host ( void* host_pointer, const void* device_pointer, int size_in_bytes )
{
	CUDA_SAFE_CALL(hipMemcpy(host_pointer, device_pointer, size_in_bytes, hipMemcpyDeviceToHost));	
}

extern "C" void free_device_memory ( void* device_pointer )
{
	CUDA_SAFE_CALL(hipFree(device_pointer));			
}

extern "C" void allocate_pinned_host_memory ( void** pointer_on_host, size_t size_in_bytes )  
{
	CUDA_SAFE_CALL(hipHostMalloc(pointer_on_host, size_in_bytes));	
}

extern "C" void free_pinned_host_memory ( void* pointer_on_host )  
{
	CUDA_SAFE_CALL(hipHostFree(pointer_on_host));	
}

extern "C" void transform_points_synchronous ( double *InputPoints, double *PointSelection, double *OutputPoints, int num_points, timingInfo_t* tInfo )
{
	#define OTHER 0
	#define CONVERT_PRE 1
	#define TO_DEVICE 2
	#define EXECUTE 3
	#define TO_HOST 4
	#define CONVERT_POST 5

    // setup the timing info struct - stores time in s
	(*tInfo).numEntries = 6;
	(*tInfo).timings = (double*)malloc ( 6*sizeof(double) );
	(*tInfo).labels = (char**)malloc ( 6*sizeof(char*) );
	for ( int i = 0 ; i < 6 ; i++ )
	{
		(*tInfo).labels[i] = (char*) malloc ( 33*sizeof(char) );
	}
	sprintf((*tInfo).labels[OTHER], "%s", "OTHER");
	sprintf((*tInfo).labels[CONVERT_PRE], "%s", "CONVERT_PRE");
	sprintf((*tInfo).labels[TO_DEVICE], "%s", "TO_DEVICE");
	sprintf((*tInfo).labels[EXECUTE], "%s", "EXECUTE");
	sprintf((*tInfo).labels[TO_HOST], "%s", "TO_HOST");
	sprintf((*tInfo).labels[CONVERT_POST], "%s", "CONVERT_POST");
	
	// use a cuda timer for timing
	unsigned int timer = 0;
	CUT_SAFE_CALL( cutCreateTimer( &timer ) );
	
	CUT_SAFE_CALL( cutStartTimer( timer));
    float *device_points;
	
	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);	
	
	// allocate pinned host memory to allow for asynchronous operations	
	float *host_points_single_p;
	allocate_pinned_host_memory ((void**)&host_points_single_p, num_points*sizeof(float)*4);
	
	dim3 threads_per_block(64, 1);
	dim3 blocks_per_grid( iDivUp(num_points, 64), 1);
	
	CUT_SAFE_CALL( cutStopTimer( timer));
    (*tInfo).timings[OTHER] = cutGetTimerValue( timer);
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	for (int point = 0; point < num_points; ++point)
	{
		int float_index = (point)*4;
		int double_index = (point)*3;
		host_points_single_p[float_index] = (float)InputPoints[double_index];
		host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
		host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
		host_points_single_p[float_index+3] = (float)PointSelection[point];
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[CONVERT_PRE] = cutGetTimerValue( timer);	
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));		
	CUDA_SAFE_CALL ( hipMemcpy(device_points, host_points_single_p, num_points*16, hipMemcpyHostToDevice) );
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[TO_DEVICE] = cutGetTimerValue( timer);	
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)(device_points), num_points);
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[EXECUTE] = cutGetTimerValue( timer);	
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	CUDA_SAFE_CALL ( hipMemcpy(host_points_single_p, device_points, num_points*16, hipMemcpyDeviceToHost) );
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[TO_HOST] = cutGetTimerValue( timer);	
	
	CUT_SAFE_CALL( cutResetTimer( timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	for (int point = 0; point < num_points; ++point)
	{
		int float_index = (point)*4;
		int double_index = (point)*3;
		OutputPoints[double_index] = host_points_single_p[float_index];
		OutputPoints[double_index+1] = host_points_single_p[float_index+1];
		OutputPoints[double_index+2] = host_points_single_p[float_index+2];
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[CONVERT_POST] = cutGetTimerValue( timer);
	
	CUT_SAFE_CALL( cutResetTimer( timer));	
	CUT_SAFE_CALL( cutStartTimer( timer));
	free_device_memory(device_points);
	free_pinned_host_memory ( host_points_single_p );
	CUT_SAFE_CALL( cutStopTimer( timer));
	(*tInfo).timings[OTHER] += cutGetTimerValue( timer);	
	CUT_SAFE_CALL ( cutDeleteTimer ( timer ));
}

extern "C" void transform_points_asynchronous ( double *InputPoints, double *PointSelection, double *OutputPoints, int num_points, timingInfo_t* tInfo )
{
	// set the number of streams
	int nstreams = 2;
	
	
    float *device_points;
	
	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);	
	
	// allocate pinned host memory to allow for asynchronous operations	
	float *host_points_single_p;
	allocate_pinned_host_memory ((void**)&host_points_single_p, num_points*sizeof(float)*4);
	
	// allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int n = 0; n < nstreams; n++)
    	CUDA_SAFE_CALL( hipStreamCreate(&(streams[n])) ); 
	
	int points_per_stream = num_points/nstreams;

	dim3 threads_per_block(32, 1);
	dim3 blocks_per_grid( iDivUp(points_per_stream, 32), 1);
	
	for ( int n = 0; n < nstreams; n++ )
	{
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			host_points_single_p[float_index] = (float)InputPoints[double_index];
			host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
			host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
			host_points_single_p[float_index+3] = (float)PointSelection[point];
		}

		CUDA_SAFE_CALL ( hipMemcpyAsync(device_points + n*points_per_stream*4, host_points_single_p + n*points_per_stream*4, points_per_stream*16, hipMemcpyHostToDevice, streams[n]) );
		linear_transform_kernel <<< blocks_per_grid, threads_per_block, 0, streams[n] >>> ((float4*)(device_points + n*points_per_stream*4), points_per_stream);
		//linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)(device_points + n*points_per_stream*16), points_per_stream);
		CUDA_SAFE_CALL ( hipMemcpyAsync(host_points_single_p + n*points_per_stream*4, device_points + n*points_per_stream*4, points_per_stream*16, hipMemcpyDeviceToHost, streams[n]) );
		
		// need to synchronize the streams so that the data is available to copy to the output points	
		hipStreamSynchronize(streams[n]);		
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			OutputPoints[double_index] = host_points_single_p[float_index];
			OutputPoints[double_index+1] = host_points_single_p[float_index+1];
			OutputPoints[double_index+2] = host_points_single_p[float_index+2];
		}
	
	}
	
	// release resources
	for(int n = 0; n < nstreams; n++)
	{
    	hipStreamDestroy(streams[n]);
	}
	
	free_device_memory(device_points);
	free_pinned_host_memory ( host_points_single_p );
}

